#include "hip/hip_runtime.h"
/* Copyright STIFTELSEN SINTEF 2012
 *
 * This file is part of the HPMC Library.
 *
 * Author(s): Christopher Dyken, <christopher.dyken@sintef.no>
 *
 * HPMC is free software: you can redistribute it and/or modify it under the
 * terms of the GNU General Public License as published by the Free Software
 * Foundation, either version 3 of the License, or (at your option) any later
 * version.
 *
 * HPMC is distributed in the hope that it will be useful, but WITHOUT ANY
 * WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS FOR
 * A PARTICULAR PURPOSE.  See the GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License along with
 * HPMC.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <iostream>
#include <stdexcept>
#include <cuhpmc/Constants.hpp>
#include <cuhpmc/FieldGlobalMemUChar.hpp>
#include <cuhpmc/IsoSurface.hpp>
#include <cuhpmc/IsoSurfaceIndexed.hpp>
#include <cuhpmc/CUDAErrorException.hpp>

namespace cuhpmc {

template<class T>
static __device__ __inline__
void
fetchFromField( uint& bp0, uint& bp1, uint& bp2, uint& bp3, uint& bp4, uint& bp5,
                const T* ptr,
                const int offset,
                const int slice_pitch,
                const float iso,
                const int chunk_cells_z )
{
    int offset_tmp = offset;

    bp0 = ptr[ offset_tmp ] < iso ? 1 : 0;

    if( 1 <= chunk_cells_z ) { offset_tmp += slice_pitch; }
    bp1 = ptr[ offset_tmp ] < iso ? 1 : 0;

    if( 2 <= chunk_cells_z ) { offset_tmp += slice_pitch; }
    bp2 = ptr[ offset_tmp ] < iso ? 1 : 0;

    if( 3 <= chunk_cells_z ) { offset_tmp += slice_pitch; }
    bp3 = ptr[ offset_tmp ] < iso ? 1 : 0;

    if( 4 <= chunk_cells_z ) { offset_tmp += slice_pitch; }
    bp4 = ptr[ offset_tmp ] < iso ? 1 : 0;

    if( 5 <= chunk_cells_z ) { offset_tmp += slice_pitch; }
    bp5 = ptr[ offset_tmp ] < iso ? 1 : 0;
}

static __device__ __inline__
void
mergeAlongY( uint& bp0, uint& bp1, uint& bp2, uint& bp3, uint& bp4,
             uint& bc0, uint& bc1, uint& bc2, uint& bc3, uint& bc4 )
{
    uint t0 = bp0 + (bc0<<2); bp0 = bc0; bc0 = t0;
    uint t1 = bp1 + (bc1<<2); bp1 = bc1; bc1 = t1;
    uint t2 = bp2 + (bc2<<2); bp2 = bc2; bc2 = t2;
    uint t3 = bp3 + (bc3<<2); bp3 = bc3; bc3 = t3;
    uint t4 = bp4 + (bc4<<2); bp4 = bc4; bc4 = t4;
}

static __device__ __inline__
void
mergeAlongZ( uint& bc0, uint& bc1, uint& bc2, uint& bc3, uint& bc4, uint& bc5 )
{
    bc0 = bc0 + (bc1<<4);
    bc1 = bc1 + (bc2<<4);
    bc2 = bc2 + (bc3<<4);
    bc3 = bc3 + (bc4<<4);
    bc4 = bc4 + (bc5<<4);
}



template<class T>
struct hp5_buildup_base_indexed_triple_gb_args
{
    uint4* __restrict__                 tri_pyramid_level_a_d;
    uint4* __restrict__                 vtx_pyramid_level_a_d;
    uint4* __restrict__                 tri_pyramid_level_b_d;
    uint4* __restrict__                 vtx_pyramid_level_b_d;
    uint4* __restrict__                 tri_pyramid_level_c_d;
    uint4* __restrict__                 vtx_pyramid_level_c_d;
    uint*  __restrict__                 tri_sideband_level_c_d;
    uint*  __restrict__                 vtx_sideband_level_c_d;
    unsigned char* __restrict__         d_case;
    float                               iso;
    int3                                cells;
    int3                                chunks;
    const T* __restrict__               field;
    int                                 field_row_pitch;
    int                                 field_slice_pitch;
    const unsigned char* __restrict__   case_vtxtricnt;
};

template<class T>
__global__
void
__launch_bounds__( 160 )
hp5_buildup_base_indexed_triple_gb( hp5_buildup_base_indexed_triple_gb_args<T> a )
{
    volatile __shared__ uint sb[800];
#if __CUDA_ARCH__ >= 300
    // we use warp shuffle instead and require less memory.
    __shared__ uint sh[160];
#else
    volatile __shared__ uint sh[800];
#endif

    const int w  = threadIdx.x / 32;                                   // warp
    const int wt = threadIdx.x % 32;                                   // thread-in-warp
    const int sh_i = 160*w + 5*wt;                                     //
    const int hp_b_o = 5*32*blockIdx.x + 32*w + wt;                    //
    const int c_lix = 5*blockIdx.x + w;                                //

    // Determine which chunk we're processing in this warp
    const int3 chunk = make_int3( c_lix % a.chunks.x,
                                  (c_lix/a.chunks.x) % a.chunks.y,
                                  (c_lix/a.chunks.x) / a.chunks.y );
    if( chunk.z < a.chunks.z ) {

        // Base xyz-index for this chunk. Can be composed into a single uint32, and
        // be checked with an LOP.AND + ISETP.LT
        const int3 chunk_offset = make_int3( 31*chunk.x,
                                             5*chunk.y,
                                             5*chunk.z );

        const int3 chunk_cells = make_int3( a.cells.x - chunk_offset.x,
                                            a.cells.y - chunk_offset.y,
                                            a.cells.z - chunk_offset.z );

        // base corner should always be inside field, but x for wt > 0 might be
        // outside.
        int field_offset = min( a.cells.z, chunk_offset.z) * a.field_slice_pitch
                         + chunk_offset.y * a.field_row_pitch
                         + chunk_offset.x + min( wt, chunk_cells.x );


        // Fetch scalar field values and determine inside-outside for 5 slices
        uint bp0, bp1, bp2, bp3, bp4, bp5;
        fetchFromField( bp0, bp1, bp2, bp3, bp4, bp5,
                        a.field, field_offset, a.field_slice_pitch, a.iso, chunk_cells.z );

        mergeAlongZ( bp0, bp1, bp2, bp3, bp4, bp5 );

        // merge along z before Y?
        for(uint q=0; q<5; q++) {
            // Move along y to build up masks
            if( q+1 <= chunk_cells.y ) {
                field_offset += a.field_row_pitch;
            }
            uint bc0, bc1, bc2, bc3, bc4, bc5;
            fetchFromField( bc0, bc1, bc2, bc3, bc4, bc5,
                            a.field, field_offset, a.field_slice_pitch, a.iso, chunk_cells.z );
            mergeAlongZ( bc0, bc1, bc2, bc3, bc4, bc5 );
            mergeAlongY( bp0, bp1, bp2, bp3, bp4,
                         bc0, bc1, bc2, bc3, bc4 );

#if __CUDA_ARCH__ >= 300
            bc0 = bc0 + ( (uint)__shfl_down( (int)bc0, 1 )<<1u);
            bc1 = bc1 + ( (uint)__shfl_down( (int)bc1, 1 )<<1u);
            bc2 = bc2 + ( (uint)__shfl_down( (int)bc2, 1 )<<1u);
            bc3 = bc3 + ( (uint)__shfl_down( (int)bc3, 1 )<<1u);
            bc4 = bc4 + ( (uint)__shfl_down( (int)bc4, 1 )<<1u);
#else
            sh[ 0*160 + threadIdx.x ] = bc0;
            sh[ 1*160 + threadIdx.x ] = bc1;
            sh[ 2*160 + threadIdx.x ] = bc2;
            sh[ 3*160 + threadIdx.x ] = bc3;
            sh[ 4*160 + threadIdx.x ] = bc4;
            if( wt < 31 ) {
                bc0 = bc0 + (sh[ 0*160 + threadIdx.x + 1]<<1);
                bc1 = bc1 + (sh[ 1*160 + threadIdx.x + 1]<<1);
                bc2 = bc2 + (sh[ 2*160 + threadIdx.x + 1]<<1);
                bc3 = bc3 + (sh[ 3*160 + threadIdx.x + 1]<<1);
                bc4 = bc4 + (sh[ 4*160 + threadIdx.x + 1]<<1);
            }
#endif
            uint ix_o_1 = 160*w + 32*q + wt;
            uint mask;
            uint sum;
            if(  (wt < 31) &&
                 (wt <= chunk_cells.x ) &&
                 (q <= chunk_cells.y ) )
            {
                if( (wt == chunk_cells.x) || (q==chunk_cells.y) ) {
                    mask = 0xf0u;
                }
                else {
                    mask = ~0x0u;
                }
            }
            else {
                mask = 0x0u;
            }
            if( mask != 0u ) {
                // cnt_a_X = %00000000 0vv00ttt
                uint cnt_a_0;
                uint cnt_a_1;
                uint cnt_a_2;
                uint cnt_a_3;
                uint cnt_a_4;
                if( mask == ~0x0u && 5 < chunk_cells.z ) {
                    cnt_a_0 = a.case_vtxtricnt[ bc0 ];
                    cnt_a_1 = a.case_vtxtricnt[ bc1 ];
                    cnt_a_2 = a.case_vtxtricnt[ bc2 ];
                    cnt_a_3 = a.case_vtxtricnt[ bc3 ];
                    cnt_a_4 = a.case_vtxtricnt[ bc4 ];
                }
                else {
                    uint tmp_mask;
                    if( 0 <= chunk_cells.z ) {
                        tmp_mask = mask;
                    }
                    else {
                        tmp_mask = 0x00u;
                    }
                    if( 0 == chunk_cells.z ) {
                        tmp_mask = tmp_mask & 0xf0u;
                        mask = 0x00u;
                    }
                    cnt_a_0 = a.case_vtxtricnt[ bc0 ] & tmp_mask;

                    tmp_mask = mask;
                    if( 1 == chunk_cells.z ) {
                        tmp_mask = tmp_mask & 0xf0u;
                        mask = 0x00u;
                    }
                    cnt_a_1 = a.case_vtxtricnt[ bc1 ] & tmp_mask;

                    tmp_mask = mask;
                    if( 2 == chunk_cells.z ) {
                        tmp_mask = tmp_mask & 0xf0u;
                        mask = 0x00u;
                    }
                    cnt_a_2 = a.case_vtxtricnt[ bc2 ] & tmp_mask;

                    tmp_mask = mask;
                    if( 3 == chunk_cells.z ) {
                        tmp_mask = tmp_mask & 0xf0u;
                        mask = 0x00u;
                    }
                    cnt_a_3 = a.case_vtxtricnt[ bc3 ] & tmp_mask;

                    tmp_mask = mask;
                    if( 4 == chunk_cells.z ) {
                        tmp_mask = tmp_mask & 0xf0u;
                    }
                    cnt_a_4 = a.case_vtxtricnt[ bc4 ] & tmp_mask;
                }
                sum = cnt_a_0
                        + cnt_a_1
                        + cnt_a_2
                        + cnt_a_3
                        + cnt_a_4;

                // sum = %00000000 00000000 0000000v vvvttttt
                // sb  = %00000000 0000vvvv 00000000 000ttttt
                sb[ ix_o_1 ] = ((sum<<11)&0xf0000u) | (sum&0x1fu);
                if( sum > 0 ) {
                    // triangle count stored as 4 x 4 bits = 16 bits
                    ((short1*)(a.tri_pyramid_level_a_d))[ 5*160*blockIdx.x + ix_o_1 ] =
                            make_short1( ((cnt_a_0 & 0xf)) |
                                         ((cnt_a_1 & 0xf)<<4) |
                                         ((cnt_a_2 & 0xf)<<8) |
                                         ((cnt_a_3 & 0xf)<<12) );

                    // vertex count stored as 4 x 2 bits = 8 bits
                    ((unsigned char*)(a.vtx_pyramid_level_a_d))[ 5*160*blockIdx.x + ix_o_1 ]
                            = ((cnt_a_0>>5u)
                               |  (cnt_a_1>>3u)
                               |  ((cnt_a_2>>1u)&0x30u)
                               |  ((cnt_a_3<<1u)&0xc0u) ) & 0xffu;


                    //   a.tri_pyramid_level_a_d[ 5*160*blockIdx.x + ix_o_1 ] = make_uint4( s0_1, s1_1, s2_1, s3_1 );
                    a.d_case[ 5*(5*160*blockIdx.x + 160*w + 32*q + wt) + 0 ] = bc0;
                    a.d_case[ 5*(5*160*blockIdx.x + 160*w + 32*q + wt) + 1 ] = bc1;
                    a.d_case[ 5*(5*160*blockIdx.x + 160*w + 32*q + wt) + 2 ] = bc2;
                    a.d_case[ 5*(5*160*blockIdx.x + 160*w + 32*q + wt) + 3 ] = bc3;
                    a.d_case[ 5*(5*160*blockIdx.x + 160*w + 32*q + wt) + 4 ] = bc4;
                }
            }
            else {
                sb[ ix_o_1 ] = 0;
            }
        }
    }
    else {
        // pad-chunk, just write zero's and we'll never see it again.
        for(uint q=0; q<5; q++) {
            uint ix_o_1 = 160*w + 32*q + wt;
            sb[ ix_o_1 ] = 0;
        }
    }
    // second reduction
    // sh_i = 160*w + 5*wt;
    uint cnt_b_0 = sb[ sh_i + 0 ];
    uint cnt_b_1 = sb[ sh_i + 1 ];
    uint cnt_b_2 = sb[ sh_i + 2 ];
    uint cnt_b_3 = sb[ sh_i + 3 ];
    uint cnt_b_4 = sb[ sh_i + 4 ];

    // triangle count as 4 x 8 bits = 32 bits
    ((uchar4*)a.tri_pyramid_level_b_d)[ hp_b_o ] = make_uchar4( cnt_b_0,
                                                                cnt_b_1,
                                                                cnt_b_2,
                                                                cnt_b_3 );

    // vertex count stored as 4 x 4 bits = 16 bits
    ((short1*)a.vtx_pyramid_level_b_d)[ hp_b_o ] = make_short1( (cnt_b_0>>16u) |
                                                                (cnt_b_1>>12u) |
                                                                (cnt_b_2>>8u)  |
                                                                (cnt_b_3>>4u) );


    __syncthreads();
    // third reduction
    // sh = %00000000 0vvvvvvv 00000000 0ttttttt
    sh[ 32*w + wt ] = cnt_b_0
                    + cnt_b_1
                    + cnt_b_2
                    + cnt_b_3
                    + cnt_b_4;
    __syncthreads();
    if( w == 0 ) {
        uint cnt_c_0 = sh[5*wt+0];
        uint cnt_c_1 = sh[5*wt+1];
        uint cnt_c_2 = sh[5*wt+2];
        uint cnt_c_3 = sh[5*wt+3];
        uint cnt_c_4 = sh[5*wt+4];

        // triangle count stored as 4 x 8 bits
        ((uchar4*)a.tri_pyramid_level_c_d)[ 32*blockIdx.x + wt ] = make_uchar4( cnt_c_0,
                                                                                cnt_c_1,
                                                                                cnt_c_2,
                                                                                cnt_c_3 );
        // vertex count stored as 4 x 8 bits
        ((uchar4*)a.vtx_pyramid_level_c_d)[ 32*blockIdx.x + wt ] = make_uchar4( (cnt_c_0>>16),
                                                                                (cnt_c_1>>16),
                                                                                (cnt_c_2>>16),
                                                                                (cnt_c_3>>16) );

        // sum = %0000000v vvvvvvvv 000000tt tttttttt
        uint sum = cnt_c_0
                 + cnt_c_1
                 + cnt_c_2
                 + cnt_c_3
                 + cnt_c_4;

        a.tri_sideband_level_c_d[ 32*blockIdx.x + wt ] = sum       & 0xffffu;
        a.vtx_sideband_level_c_d[ 32*blockIdx.x + wt ] = (sum>>16) & 0xffffu;
    }
}

void
IsoSurfaceIndexed::invokeBaseBuildup( hipStream_t stream )
{
    if( FieldGlobalMemUChar* field = dynamic_cast<FieldGlobalMemUChar*>( m_field ) ) {

        hp5_buildup_base_indexed_triple_gb_args<unsigned char> args;
        args.tri_pyramid_level_a_d  = m_triangle_pyramid_d + m_hp5_offsets[ m_hp5_levels-1 ];
        args.vtx_pyramid_level_a_d  = m_vertex_pyramid_d   + m_hp5_offsets[ m_hp5_levels-1 ];
        args.tri_pyramid_level_b_d  = m_triangle_pyramid_d + m_hp5_offsets[ m_hp5_levels-2 ];
        args.vtx_pyramid_level_b_d  = m_vertex_pyramid_d   + m_hp5_offsets[ m_hp5_levels-2 ];
        args.tri_pyramid_level_c_d  = m_triangle_pyramid_d + m_hp5_offsets[ m_hp5_levels-3 ];
        args.vtx_pyramid_level_c_d  = m_vertex_pyramid_d   + m_hp5_offsets[ m_hp5_levels-3 ];
        args.tri_sideband_level_c_d = m_triangle_sideband_d + m_hp5_offsets[ m_hp5_levels-3 ];
        args.vtx_sideband_level_c_d = m_vertex_sideband_d + m_hp5_offsets[ m_hp5_levels-3 ];
        args.d_case             = m_case_d;
        args.iso                = 256.f*m_iso;
        args.cells              = make_int3( field->width()-1,
                                             field->height()-1,
                                             field->depth()-1 );
        args.chunks             = make_int3( m_hp5_chunks.x,
                                             m_hp5_chunks.y,
                                             m_hp5_chunks.z );
        args.field              = field->fieldDev();
        args.field_row_pitch    = field->width();
        args.field_slice_pitch  = field->width()*field->height();
        args.case_vtxtricnt     = m_constants->vertexTriangleCountDev() ;

        uint gs = (m_hp5_level_sizes[ m_hp5_levels-1 ]+3999)/4000;
        uint bs = 160;
        hp5_buildup_base_indexed_triple_gb<unsigned char><<<gs,bs,0, stream >>>( args );

    }
    else {
        throw std::runtime_error( "invokeBaseBuildup: unsupported field type" );
    }
    hipError_t error = hipGetLastError();
    if( error != hipSuccess ) {
        throw CUDAErrorException( error );
    }
}


} // of namespace cuhpmc
