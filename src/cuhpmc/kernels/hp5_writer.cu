#include "hip/hip_runtime.h"
/* Copyright STIFTELSEN SINTEF 2012
 *
 * This file is part of the HPMC Library.
 *
 * Author(s): Christopher Dyken, <christopher.dyken@sintef.no>
 *
 * HPMC is free software: you can redistribute it and/or modify it under the
 * terms of the GNU General Public License as published by the Free Software
 * Foundation, either version 3 of the License, or (at your option) any later
 * version.
 *
 * HPMC is distributed in the hope that it will be useful, but WITHOUT ANY
 * WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS FOR
 * A PARTICULAR PURPOSE.  See the GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License along with
 * HPMC.  If not, see <http://www.gnu.org/licenses/>.
 */
#include <iostream>
#include <hip/hip_runtime.h>
#include <builtin_types.h>

namespace cuhpmc {

// constant mem size: 64kb, cache working set: 8kb.
// Count + pad :  1+3 elements :    16 bytes :    16 bytes
// Level 0     :    4 elements :    16 bytes :    32 bytes
// Level 1     :   20 elements :    80 bytes :   112 bytes
// Level 2     :  100 elements :   400 bytes :   512 bytes
// Level 3     :  500 elements :  2000 bytes :  2112 bytes
// Level 4     : 2500 elements : 10000 bytes : 12112 bytes
// Levels 0-2: 32*4*4=512 bytes :
// Level  3:

texture<uint4, 1, hipReadModeElementType> hp5_hp_tex;
__constant__ uint4 hp5_hp_const[528]; // = 2112/4
__constant__ uint  hp5_const_offsets[32];


__device__
void
downTraversalStep( uint& pos, uint& key, const uint4& val )
{
    pos *= 5;
    if( val.x <= key ) {
        pos++;
        key -=val.x;
        if( val.y <= key ) {
            pos++;
            key-=val.y;
            if( val.z <= key ) {
                pos++;
                key-=val.z;
                if( val.w <= key ) {
                    pos++;
                    key-=val.w;
                }
            }
        }
    }
}

__device__
void
hp5PosToCellPos( uint3&                             i0,
                 uint&                              mc_case,
                 const uint                         pos,
                 const uint3&                       chunks,
                 const unsigned char* __restrict__  mc_cases_d )
{
    uint c_lix = pos / 800u;
    uint t_lix = pos % 800u;
    uint3 ci = make_uint3( 31*( c_lix % chunks.x ),
                           5*( (c_lix/chunks.x) % chunks.y ),
                           5*( (c_lix/chunks.x) / chunks.y ) );

    // calc 3D pos within cunk
    i0 = make_uint3( ci.x + ((t_lix / 5)%32),
                     ci.y + ((t_lix / 5)/32),
                     ci.z + ( t_lix%5 ) );

    mc_case = mc_cases_d[ pos ];
}



template<bool use_texfetch,bool use_constmem>
__global__
void
dummy_writer( float* __restrict__               output_d,
              const uint4* __restrict__         hp5_d,
              const unsigned char* __restrict__ mc_cases_d,
              const unsigned char* __restrict__ case_intersect_edge_d,
              const uint3                       chunks,
              const uint                        triangles,
              const uint                        max_level,
              const float                       iso,
              const unsigned char*              field_d,
              const uint                        field_row_pitch,
              const uint                        field_slice_pitch,
              const float3                      scale )
{

    uint triangle = 256*blockIdx.x + threadIdx.x;

    if( triangle < triangles ) {

        uint key = triangle;
        uint pos = 0;
        int l = 0;
        if( use_constmem ) {
            for(l=0; l<4; l++ ) {
                uint4 val = hp5_hp_const[ hp5_const_offsets[l] + pos ];
                downTraversalStep( pos, key, val );
            }
        }
        for(; l<max_level; l++) {
            uint4 val;
            if(use_texfetch) {
                val = tex1Dfetch( hp5_hp_tex, hp5_const_offsets[l] + pos );
            }
            else {
                val = hp5_d[ hp5_const_offsets[l] + pos ];
            }
            downTraversalStep( pos, key, val );
        }
        uint rem = 3*key;

        uint3 i0;
        uint mc_case;
        hp5PosToCellPos( i0, mc_case, pos, chunks, mc_cases_d );
        for(uint i=0; i<3; i++ ) {
            uint isec = case_intersect_edge_d[ 16*mc_case + rem + i ];

            uint3 oa = make_uint3( i0.x + ((isec   )&1u),
                                   i0.y + ((isec>>1u)&1u),
                                   i0.z + ((isec>>2u)&1u) );
            uint oa_ix = oa.x
                       + oa.y*field_row_pitch
                       + oa.z*field_slice_pitch;
            float fa = field_d[ oa_ix ];
            float fa_x = field_d[ oa_ix + 1 ]-fa;
            float fa_y = field_d[ oa_ix + field_row_pitch ]-fa;
            float fa_z = field_d[ oa_ix + field_slice_pitch ]-fa;

            uint3 ob = make_uint3( i0.x + ((isec>>3u)&1u),
                                   i0.y + ((isec>>4u)&1u),
                                   i0.z + ((isec>>5u)&1u) );
            uint ob_ix = ob.x
                       + ob.y*field_row_pitch
                       + ob.z*field_slice_pitch;
            float fb = field_d[ ob_ix ];
            float fb_x = field_d[ ob_ix + 1 ]-fb;
            float fb_y = field_d[ ob_ix + field_row_pitch ]-fb;
            float fb_z = field_d[ ob_ix + field_slice_pitch ]-fb;

            float t = (iso-fa)/(fb-fa);
            float s = 1.f-t;

            float n_x = s*fa_x + t*fb_x;
            float n_y = s*fa_y + t*fb_y;
            float n_z = s*fa_z + t*fb_z;


            uint vtx = 3*triangle + i;

            output_d[ 6*vtx + 0 ] = n_x;
            output_d[ 6*vtx + 1 ] = n_y;
            output_d[ 6*vtx + 2 ] = n_z;
            output_d[ 6*vtx + 3 ] = scale.x*(s*oa.x + t*ob.x);
            output_d[ 6*vtx + 4 ] = scale.y*(s*oa.y + t*ob.y);
            output_d[ 6*vtx + 5 ] = scale.z*(s*oa.z + t*ob.z);
        }
    }
}




void
run_dummy_writer( float*                output_d,
                  const uint4*          hp5_pyramid_d,
                  const unsigned char*  mc_cases_d,
                  const unsigned char*  case_intersect_edge_d,
                  const uint*           hp5_level_offsets_d,
                  const uint3           hp5_chunks,
                  const uint            hp5_size,
                  const uint            hp5_max_level,
                  const uint            vertices,
                  const float           iso,
                  const unsigned char*  field_d,
                  const uint3           field_size,
                  hipStream_t          stream )
{
    if( vertices == 0 ) {
        return;
    }
    const uint triangles = vertices/3;

    bool use_constmem = true;
    bool use_texfetch = true;

    // Copy offsets to symbol
    hipMemcpyToSymbolAsync(HIP_SYMBOL( hp5_const_offsets),
                             hp5_level_offsets_d,
                             sizeof(uint)*32,
                             0,
                             hipMemcpyDeviceToDevice,
                             stream );
    // Copy top levels of hp if desired
    if( use_constmem ) {
        hipMemcpyToSymbolAsync(HIP_SYMBOL( hp5_hp_const),
                                 hp5_pyramid_d,
                                 528*sizeof(uint4),
                                 0,
                                 hipMemcpyDeviceToDevice,
                                 stream );
    }
    // Bind histopyramid as texture if desired
    if( use_texfetch ) {
        hipBindTexture( NULL,
                         hp5_hp_tex,
                         hp5_pyramid_d,
                         hipCreateChannelDesc( 32, 32, 32, 32,
                                                hipChannelFormatKindUnsigned ),
                         4*sizeof(uint)*hp5_size );
    }


    dim3 gs( ((triangles+255)/256), 1, 1 );
    dim3 bs( 256, 1, 1 );


    if( use_texfetch ) {
        if( use_constmem ) {
            dummy_writer<true,true><<<gs,bs,0,stream>>>( output_d,
                                                         hp5_pyramid_d,
                                                         mc_cases_d,
                                                         case_intersect_edge_d,
                                                         hp5_chunks,
                                                         triangles,
                                                         hp5_max_level,
                                                         256.f*iso,
                                                         field_d,
                                                         field_size.x,
                                                         field_size.x*field_size.y,
                                                         make_float3( 1.f/(field_size.x-1.f),
                                                                      1.f/(field_size.y-1.f),
                                                                      1.f/(field_size.z-1.f) ) );
        }
        else {
            dummy_writer<true,false><<<gs,bs,0,stream>>>( output_d,
                                                          hp5_pyramid_d,
                                                          mc_cases_d,
                                                          case_intersect_edge_d,
                                                          hp5_chunks,
                                                          triangles,
                                                          hp5_max_level,
                                                          256.f*iso,
                                                          field_d,
                                                          field_size.x,
                                                          field_size.x*field_size.y,
                                                          make_float3( 1.f/(field_size.x-1.f),
                                                                       1.f/(field_size.y-1.f),
                                                                       1.f/(field_size.z-1.f) ) );
        }
    }
    else {
        if( use_constmem ) {
            dummy_writer<false,true><<<gs,bs,0,stream>>>( output_d,
                                                          hp5_pyramid_d,
                                                          mc_cases_d,
                                                          case_intersect_edge_d,
                                                          hp5_chunks,
                                                          triangles,
                                                          hp5_max_level,
                                                          256.f*iso,
                                                          field_d,
                                                          field_size.x,
                                                          field_size.x*field_size.y,
                                                          make_float3( 1.f/(field_size.x-1.f),
                                                                       1.f/(field_size.y-1.f),
                                                                       1.f/(field_size.z-1.f) ) );
        }
        else {
            dummy_writer<false,false><<<gs,bs,0,stream>>>( output_d,
                                                           hp5_pyramid_d,
                                                           mc_cases_d,
                                                           case_intersect_edge_d,
                                                           hp5_chunks,
                                                           triangles,
                                                           hp5_max_level,
                                                           256.f*iso,
                                                           field_d,
                                                           field_size.x,
                                                           field_size.x*field_size.y,
                                                           make_float3( 1.f/(field_size.x-1.f),
                                                                        1.f/(field_size.y-1.f),
                                                                        1.f/(field_size.z-1.f) ) );
        }
    }
}



} // of namespace cuhpmc
