#include "hip/hip_runtime.h"
/* Copyright STIFTELSEN SINTEF 2012
 *
 * This file is part of the HPMC Library.
 *
 * Author(s): Christopher Dyken, <christopher.dyken@sintef.no>
 *
 * HPMC is free software: you can redistribute it and/or modify it under the
 * terms of the GNU General Public License as published by the Free Software
 * Foundation, either version 3 of the License, or (at your option) any later
 * version.
 *
 * HPMC is distributed in the hope that it will be useful, but WITHOUT ANY
 * WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS FOR
 * A PARTICULAR PURPOSE.  See the GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License along with
 * HPMC.  If not, see <http://www.gnu.org/licenses/>.
 */
#include <GL/glew.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <cuda_gl_interop.h>
#include <builtin_types.h>
#include <sstream>
#include <vector>
#include <iomanip>
#include <iostream>
#include "../common/common.hpp"
#include <cuhpmc/Constants.hpp>
#include <cuhpmc/FieldGlobalMemUChar.hpp>
#include <cuhpmc/GLFieldUCharBuffer.hpp>
#include <cuhpmc/IsoSurface.hpp>
#include <cuhpmc/GLIsoSurface.hpp>
#include <cuhpmc/TriangleVertexWriter.hpp>
#include <cuhpmc/GLWriter.hpp>

using std::cerr;
using std::endl;

int                             volume_size_x       = 128;
int                             volume_size_y       = 128;
int                             volume_size_z       = 128;
float                           iso                 = 0.5f;

cuhpmc::Constants*              constants           = NULL;
unsigned char*                  field_data_dev      = NULL;
cuhpmc::AbstractField*          field               = NULL;
cuhpmc::AbstractIsoSurface*     iso_surface         = NULL;
cuhpmc::AbstractWriter*         writer              = NULL;

GLuint                          surface_vao         = 0;
GLuint                          surface_vbo         = 0;
GLsizei                         surface_vbo_n       = 0;
hipGraphicsResource*           surface_resource    = NULL;
hipStream_t                    stream              = 0;
float*                          surface_cuda_d      = NULL;
hipEvent_t                     pre_buildup         = 0;
hipEvent_t                     post_buildup        = 0;
float                           buildup_ms          = 0.f;

hipEvent_t                     pre_write           = 0;
hipEvent_t                     post_write          = 0;
float                           write_ms            = 0.f;

uint                            runs                = 0;

bool                            profile             = false;
bool                            gl_direct_draw      = true;
GLuint                          gl_field_buffer     = 0;

template<class type, bool clamp, bool half_float>
__global__
void
bumpyCayley( type* __restrict__ output,
             const uint field_x,
             const uint field_y,
             const uint field_z,
             const uint field_row_pitch,
             const uint field_slice_pitch,
             const uint by )
{
    uint ix = blockIdx.x*blockDim.x + threadIdx.x;
    uint iy = (blockIdx.y%by)*blockDim.y + threadIdx.y;
    uint iz = blockIdx.y/by;
    if( ix < field_x && iy < field_y && iz < field_z ) {
        float x = 2.f*(float)ix/(float)field_x-1.f;
        float y = 2.f*(float)iy/(float)field_y-1.f;
        float z = 2.f*(float)iz/(float)field_z-1.f;
        float f = 16.f*x*y*z + 4.f*x*x + 4.f*y*y + 4.f*z*z - 1.f;
//                  + 0.2f*sinf(33.f*x)*cosf(39.1f*y)*sinf(37.3f*z)
//                  + 0.1f*sinf(75.f*x)*cosf(75.1f*y)*sinf(71.3f*z);
/*
        - 0.6*sinf(25.1*x)*cosf(23.2*y)*sinf(21*z)
                  + 0.4*sinf(41.1*x*y)*cosf(47.2*y)*sinf(45*z)
                  - 0.2*sinf(111.1*x*y)*cosf(117.2*y)*sinf(115*z);
*/
//        f = sin(f);
        if( clamp ) {
            f = 255.f*f;
            if( f > 255 ) f = 255.f;
            if( f < 0 ) f = 0.f;
        }
        if( half_float ) {
            output[ ix + iy*field_row_pitch + iz*field_slice_pitch ] = __float2half_rn( f );
        }
        else {
            output[ ix + iy*field_row_pitch + iz*field_slice_pitch ] = f;
        }
    }
}


void
printHelp( const std::string& appname )
{
    //       --------------------------------------------------------------------------------
    cerr << "HPMC demo application that visualizes 1-16xyz-4x^2-4y^2-4z^2=iso."<<endl<<endl;
    cerr << "Usage: " << appname << " [options] xsize [ysize zsize] "<<endl<<endl;
    cerr << "where: xsize    The number of samples in the x-direction."<<endl;
    cerr << "       ysize    The number of samples in the y-direction."<<endl;
    cerr << "       zsize    The number of samples in the z-direction."<<endl;
    cerr << "Example usage:"<<endl;
    cerr << "    " << appname << " 64"<< endl;
    cerr << "    " << appname << " 64 128 64"<< endl;
    cerr << endl;
    cerr << "Options specific for this app:" << std::endl;
    cerr << "    --device <int>  Specify which CUDA device to use." << endl;
    cerr << "    --gl-direct     Enable direct rendering in OpenGL (i.e., do not store" << endl;
    cerr << "                    geomtry in a buffer)." << endl;
    cerr << "    --no-gl-direct  Disable direct rendering in OpenGL (i.e., geometry is stored" << endl;
    cerr << "                    in a buffer by CUDA, which OpenGL then renders)." << endl;
    cerr << "    --profile       Enable profiling of CUDA passes." << endl;
    cerr << "    --no-profile    Disable profiling of CUDA passes." << endl;
    cerr << endl;
    printOptions();
}


void
init( int argc, char** argv )
{
    int device = -1;
    for( int i=1; i<argc; ) {
        int eat = 0;
        std::string arg( argv[i] );
        if( (arg == "--device") && (i+1)<argc ) {
            device = atoi( argv[i+1] );
            eat = 2;
        }
        else if( (arg == "--gl-direct" ) ) {
            gl_direct_draw = true;
            eat = 1;
        }
        else if( (arg == "--no-gl-direct" ) ) {
            gl_direct_draw = false;
            eat = 1;
        }
        else if( (arg == "--profile" ) ) {
            profile = true;
            eat = 1;
        }
        else if( (arg == "--no-profile" ) ) {
            profile = false;
            eat = 1;
        }

        if( eat ) {
            argc = argc - eat;
            for( int k=i; k<argc; k++ ) {
                argv[k] = argv[k+eat];
            }
        }
        else {
            i++;
        }
    }
    if( argc > 1 ) {
        volume_size_x = atoi( argv[1] );
    }
    if( argc > 3 ) {
        volume_size_y = atoi( argv[2] );
        volume_size_z = atoi( argv[3] );
    }
    else {
        volume_size_y = volume_size_x;
        volume_size_z = volume_size_x;
    }
    if( volume_size_x < 4 ) {
        cerr << "Volume size x < 4" << endl;
        exit( EXIT_FAILURE );
    }
    if( volume_size_y < 4 ) {
        cerr << "Volume size y < 4" << endl;
        exit( EXIT_FAILURE );
    }
    if( volume_size_z < 4 ) {
        cerr << "Volume size z < 4" << endl;
        exit( EXIT_FAILURE );
    }

    int device_n = 0;
    hipGetDeviceCount( &device_n );
    if( device_n == 0 ) {
        std::cerr << "Found no CUDA capable devices." << endl;
        exit( EXIT_FAILURE );
    }
    std::cerr << "Found " << device_n << " CUDA enabled devices:" << endl;
    int best_device = -1;
    int best_device_major = -1;
    int best_device_minor = -1;
    for(int i=0; i<device_n; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties( &prop, i );
        if( (prop.major > best_device_major) || ( (prop.major==best_device_major)&&(prop.minor>best_device_minor) ) ) {
            best_device = i;
            best_device_major = prop.major;
            best_device_minor = prop.minor;
        }
        std::cerr << "    device " << i
                  << ": compute cap=" << prop.major << "." << prop.minor
                  << endl;
    }
    if( device < 0 ) {
        std::cerr << "No CUDA device specified, using device " << best_device << endl;
        device = best_device;
    }
    if( (device < 0) || (device_n <= device) ) {
        std::cerr << "Illegal CUDA device " << device << endl;
        exit( EXIT_FAILURE );
    }
    cudaGLSetGLDevice( device );
//    hipSetDevice( device );

    // create field

    hipMalloc( (void**)&field_data_dev, sizeof(unsigned char)*volume_size_x*volume_size_y*volume_size_z );
    bumpyCayley<unsigned char, true, false>
            <<< dim3( (volume_size_x+31)/32, volume_size_z*((volume_size_y+31)/32)), dim3(32,32) >>>
            ( field_data_dev,
              volume_size_x,
              volume_size_y,
              volume_size_z,
              volume_size_x,
              volume_size_x*volume_size_y,
              (volume_size_y+31)/32 );

    std::vector<unsigned char> moo( volume_size_x*volume_size_y*volume_size_z );
    hipMemcpy( moo.data(), field_data_dev, moo.size(), hipMemcpyDeviceToHost );

    // Generate OpenGL VBO that we lend to CUDA
    surface_vbo_n = 100;
    glGenBuffers( 1, &surface_vbo );
    glBindBuffer( GL_ARRAY_BUFFER, surface_vbo );
    glBufferData( GL_ARRAY_BUFFER,
                  3*2*3*sizeof(GLfloat)*surface_vbo_n,
                  NULL,
                  GL_DYNAMIC_COPY );
    glGenVertexArrays( 1, &surface_vao );
    glBindVertexArray( surface_vao );
    glVertexAttribPointer( 0, 3, GL_FLOAT, GL_FALSE, sizeof(GLfloat)*6, (void*)(3*sizeof(GLfloat)) );
    glVertexAttribPointer( 1, 3, GL_FLOAT, GL_FALSE, sizeof(GLfloat)*6, NULL );
    glEnableVertexAttribArray(0);
    glEnableVertexAttribArray(1);
    glBindVertexArray( 0);
    glBindBuffer( GL_ARRAY_BUFFER, 0 );

    hipStreamCreate( &stream );

    // Create profiling events if needed
    if( profile ) {
        hipEventCreate( &pre_buildup );
        hipEventCreate( &post_buildup );
        hipEventCreate( &pre_write );
        hipEventCreate( &post_write );
    }

    hipGraphicsGLRegisterBuffer( &surface_resource,
                                  surface_vbo,
                                  hipGraphicsRegisterFlagsWriteDiscard );


    constants = new cuhpmc::Constants();
    if( gl_direct_draw ) {
        std::vector<unsigned char> foo( volume_size_x*volume_size_y*volume_size_z );
        hipMemcpy( foo.data(), field_data_dev, foo.size(), hipMemcpyDeviceToHost );
        hipFree( field_data_dev );
        field_data_dev = NULL;

        glGenBuffers( 1, &gl_field_buffer );
        glBindBuffer( GL_TEXTURE_BUFFER, gl_field_buffer );
        glBufferData( GL_TEXTURE_BUFFER,
                      foo.size(),
                      foo.data(),
                      GL_STATIC_DRAW );
        glBindBuffer( GL_TEXTURE_BUFFER, 0 );

        field = new cuhpmc::GLFieldUCharBuffer( constants,
                                                gl_field_buffer,
                                                volume_size_x,
                                                volume_size_y,
                                                volume_size_z );

        cuhpmc::GLIsoSurface* srf = new cuhpmc::GLIsoSurface( field );
        iso_surface = srf;
        writer = new cuhpmc::GLWriter( srf );
    }
    else {
        field = new cuhpmc::FieldGlobalMemUChar( constants,
                                                 field_data_dev,
                                                 volume_size_x,
                                                 volume_size_y,
                                                 volume_size_z );
        iso_surface = new cuhpmc::IsoSurface( field );
        writer = new cuhpmc::TriangleVertexWriter( iso_surface );
    }


    hipError_t error = hipGetLastError();
    if( error != hipSuccess ) {
        std::cerr << "CUDA error: " << hipGetErrorString( error ) << endl;
        exit( EXIT_FAILURE );
    }
}



void
render( float t,
        float dt,
        float fps,
        const GLfloat* P,
        const GLfloat* MV,
        const GLfloat* PM,
        const GLfloat *NM,
        const GLfloat* MV_inv )
{
    glClear( GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT );
    glEnable( GL_DEPTH_TEST );

    iso = 0.5f;//*(sin(t)+1.f);

    // build histopyramid
    if( profile ) {
        hipEventRecord( pre_buildup, stream );
    }
    iso_surface->build( iso, stream );
    if( profile ) {
        hipEventRecord( post_buildup, stream );
    }

    // resize buffers if we run unless we do direct GL rendering
    uint triangles = 0;
    if( !gl_direct_draw ) {

        triangles = iso_surface->triangles();
        if( surface_vbo_n < triangles ) {
            if( hipGraphicsUnregisterResource( surface_resource ) == hipSuccess ) {
                if( surface_cuda_d != NULL ) {
                    hipFree( surface_cuda_d );
                    surface_cuda_d = NULL;
                }

                surface_vbo_n = 1.1f*triangles;

                std::vector<GLfloat> foo( 6*3*surface_vbo_n, 0.25f );
                for(size_t i=0; i<3*surface_vbo_n; i++ ) {
                    foo[6*i+3] = 0.5f*(cos( 0.1f*i )+1.f);
                    foo[6*i+4] = 0.5f*(cos( 0.2f*i )+1.f);
                    foo[6*i+5] = 0.5f*(cos( 0.3f*i )+1.f);
                }

                glBindBuffer( GL_ARRAY_BUFFER, surface_vbo );
                glBindBuffer( GL_ARRAY_BUFFER, surface_vbo );
                glBufferData( GL_ARRAY_BUFFER,
                              3*2*3*sizeof(GLfloat)*surface_vbo_n,
                              foo.data(),
                              GL_DYNAMIC_COPY );
                glBindBuffer( GL_ARRAY_BUFFER, surface_vbo );

                hipGraphicsGLRegisterBuffer( &surface_resource,
                                              surface_vbo,
                                              hipGraphicsRegisterFlagsNone ) /*,
                                              hipGraphicsRegisterFlagsWriteDiscard )*/;
            }
            std::cerr << "Resized VBO to hold " << triangles << " triangles (" << (3*2*3*sizeof(GLfloat)*surface_vbo_n) << " bytes)\n";
        }
    }

    if( profile ) {
        hipEventRecord( pre_write );
    }

    // direct rendering through OpenGL
    if( gl_direct_draw ) {

        if( cuhpmc::GLWriter* w = dynamic_cast<cuhpmc::GLWriter*>( writer ) ) {
            w->render( PM, NM, stream );
        }

    }
    // Let CUDA write, but don't use interop (i.e., no rendering)
    else if( wireframe ) {
        if( cuhpmc::TriangleVertexWriter* w = dynamic_cast<cuhpmc::TriangleVertexWriter*> ( writer ) ) {
            if( surface_cuda_d == NULL ) {
                hipMalloc( &surface_cuda_d, 3*2*3*sizeof(GLfloat)*surface_vbo_n );
            }
            w->writeInterleavedNormalPosition( surface_cuda_d, triangles, stream );
        }
    }
    // Let CUDA write and let GL render the resulting buffer
    else {
        if( cuhpmc::TriangleVertexWriter* w = dynamic_cast<cuhpmc::TriangleVertexWriter*> ( writer ) ) {
            if( hipGraphicsMapResources( 1, &surface_resource, stream ) == hipSuccess ) {
                float* surface_d = NULL;
                size_t surface_size = 0;
                if( hipGraphicsResourceGetMappedPointer( (void**)&surface_d,
                                                          &surface_size,
                                                          surface_resource ) == hipSuccess )
                {
                    w->writeInterleavedNormalPosition( surface_d, triangles, stream );
                }
                hipGraphicsUnmapResources( 1, &surface_resource, stream );
            }
            glMatrixMode( GL_PROJECTION );
            glLoadMatrixf( P );
            glMatrixMode( GL_MODELVIEW );
            glLoadMatrixf( MV );

            glBindVertexArray( surface_vao );
            glDrawArrays( GL_POINTS, 0, 3*triangles );
            glBindVertexArray( 0 );
        }
    }

    if( profile ) {
        hipEventRecord( post_write );
        hipEventSynchronize( post_write );

        float ms;
        hipEventElapsedTime( &ms, pre_buildup, post_buildup );
        buildup_ms += ms;
        hipEventElapsedTime( &ms, pre_write, post_write );
        write_ms += ms;
        runs++;
    }

    hipError_t error = hipGetLastError();
    if( error != hipSuccess ) {
        std::cerr << "CUDA error: " << hipGetErrorString( error ) << endl;
        exit( EXIT_FAILURE );
    }
}

const std::string
infoString( float fps )
{
    float avg_buildup = buildup_ms/runs;
    float avg_write = write_ms/runs;
    buildup_ms = 0.f;
    write_ms = 0.f;
    runs = 0;


    std::stringstream o;
    o << std::setprecision(5) << fps << " fps, "
      << volume_size_x << 'x'
      << volume_size_y << 'x'
      << volume_size_z << " samples, "
      << (int)( ((volume_size_x-1)*(volume_size_y-1)*(volume_size_z-1)*fps)/1e6 )
      << " MVPS, ";
    if( profile ) {
        o << "build=" << avg_buildup << "ms, "
          << "write=" << avg_write << "ms, ";
    }
    o << iso_surface->triangles()
      << " triangles, iso=" << iso
      << (wireframe?"[wireframe]":"");
    return o.str();
}
